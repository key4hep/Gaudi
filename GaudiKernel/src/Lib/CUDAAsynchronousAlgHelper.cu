#include <Gaudi/AsynchronousAlgorithm.h>

#include <boost/fiber/cuda/waitfor.hpp>

#include <atomic>
#include <cstdio>
#include <memory_resource>
#include <string>
#include <tuple>

#include <boost/lockfree/queue.hpp>
#include <vecmem/memory/binary_page_memory_resource.hpp>

#include <fmt/format.h>

namespace Gaudi {
  namespace CUDA {
    std::atomic_int running_streams = 0;

    class StreamList {
    private:
      boost::lockfree::queue<hipStream_t> queue{ 5 };

    public:
      boost::lockfree::queue<hipStream_t>* operator->() { return &queue; }
      ~StreamList() {
        while ( !queue.empty() ) {
          hipStream_t stream = nullptr;
          if ( queue.pop( stream ) ) hipStreamDestroy( stream );
        }
      }
    };
    StreamList  available_streams{};
    std::string err_fmt( hipError_t err, std::string file, int line ) {
      const char* errname = hipGetErrorName( err );
      const char* errstr  = hipGetErrorString( err );
      std::string errmsg =
          fmt::format( "Encountered CUDA error {} [{}]: {} on {}:{}", errname, int( err ), errstr, file, line );
      return errmsg;
    }

    hipError_t cuda_stream_await( hipStream_t cudaStream ) {
      auto        res = boost::fibers::cuda::waitfor_all( cudaStream );
      hipError_t err = std::get<1>( res );
      return err;
    }

    namespace Detail {
      void* PinnedMemoryResource::do_allocate( std::size_t bytes, std::size_t /* alignment */ ) {
        hipError_t firstErr = hipGetLastError();
        if ( firstErr != hipSuccess ) {
          // throw GaudiException(
          //     fmt::format( "First error {} ({}): {}", hipGetErrorName( firstErr ), int( firstErr ), hipGetErrorString( firstErr ) ),
          //     "CUDA_ERROR", StatusCode::FAILURE );
        }
        num_allocs.fetch_add( 1 );
        void*       ptr = nullptr;
        hipError_t err = hipHostMalloc( &ptr, bytes );
        if ( err == hipSuccess ) { return ptr; }
        if ( err == hipErrorInvalidValue || err == hipErrorOutOfMemory ) {
          throw std::bad_alloc();
        } else {
          throw GaudiException( fmt::format( "in hipHostMalloc({}, /**/) {} ({}): {}", bytes, hipGetErrorName( err ),
                                             int( err ), hipGetErrorString( err ) ),
                                "CUDA_ERROR", StatusCode::FAILURE );
        }
      }

      void PinnedMemoryResource::do_deallocate( void* p, std::size_t bytes, std::size_t /* alignment */ ) {
        num_deallocs.fetch_add( 1 );
        hipError_t err = hipHostFree( p );
        if ( err != hipSuccess && err != hipErrorDeinitialized ) {
          throw GaudiException( fmt::format( "CUDA ERROR {}: {}", hipGetErrorName( err ), hipGetErrorString( err ) ),
                                "CUDA_ERROR", StatusCode::FAILURE );
        }
      }

      bool PinnedMemoryResource::do_is_equal( const std::pmr::memory_resource& other ) const noexcept {
        return bool( dynamic_cast<const PinnedMemoryResource*>( &other ) );
      }
    } // namespace Detail

    std::pmr::memory_resource* get_pinned_memory_resource() {
      fmt::print( "Initializing pinned memory resource\n" );
      static auto base = std::make_unique<Detail::PinnedMemoryResource>();
      static auto bpmr = std::make_unique<vecmem::binary_page_memory_resource>( *base );
      static auto res  = std::make_unique<std::pmr::synchronized_pool_resource>( bpmr.get() );
      return dynamic_cast<std::pmr::memory_resource*>( res.get() );
    }

    CUDAStream::CUDAStream( const Gaudi::AsynchronousAlgorithm* parent, std::string file, int line )
        : stream( nullptr ), parent( parent ) {
      nth_stream = running_streams.fetch_add( 1 ) + 1;
      fmt::print( "Starting {}th concurrent stream\n", nth_stream );
      if ( !available_streams->pop( stream ) ) {
        hipError_t err = hipStreamCreate( &stream );
        if ( err != hipSuccess ) { parent->print_cuda_error( err_fmt( err, __FILE__, __LINE__ ) ); }
      }
    }

    CUDAStream::~CUDAStream() {
      for ( void* allocation : allocations ) { hipFreeAsync( allocation, stream ); }
      if ( parent->cuda_stream_await( stream ).isFailure() ) {
        parent->print_cuda_error( "Error freeing CUDA stream memory" );
      }
      running_streams.fetch_sub( 1 );
      available_streams->push( stream );
    }
  } // namespace CUDA
} // namespace Gaudi
