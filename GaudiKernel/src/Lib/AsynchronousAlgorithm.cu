/***********************************************************************************\
* (c) Copyright 2023-2024 CERN for the benefit of the LHCb and ATLAS collaborations *
*                                                                                   *
* This software is distributed under the terms of the Apache version 2 licence,     *
* copied verbatim in the file "LICENSE".                                            *
*                                                                                   *
* In applying this licence, CERN does not waive the privileges and immunities       *
* granted to it by virtue of its status as an Intergovernmental Organization        *
* or submit itself to any jurisdiction.                                             *
\***********************************************************************************/
#include <Gaudi/CUDAAsynchronousAlgHelper.cuh>
#include <boost/fiber/cuda/waitfor.hpp>

#include <cstdio>
#include <cstring>
#include <tuple>

namespace Gaudi {
  namespace CUDA {
    const char* err_fmt( hipError_t err ) {
      if ( err == hipSuccess ) {
        return nullptr;
      } else {
        const char* errname = hipGetErrorName( err );
        const char* errstr  = hipGetErrorString( err );
        std::size_t msg_len = 28 + std::strlen( errname ) + std::strlen( errstr );
        char*       buf     = new char[msg_len];
        std::snprintf( buf, msg_len, "Encountered CUDA error %s: %s\n", errname, errstr );
        return buf;
      }
    }

    const char* create_stream( hipStream_t* stream_ptr ) { return err_fmt( hipStreamCreate( stream_ptr ) ); }

    const char* delete_stream( hipStream_t stream ) { return err_fmt( hipStreamDestroy( stream ) ); }

    const char* cuda_stream_await( hipStream_t cudaStream ) {
      auto        res = boost::fibers::cuda::waitfor_all( cudaStream );
      hipError_t err = std::get<1>( res );
      return err_fmt( err );
    }

  } // namespace CUDA
} // namespace Gaudi
