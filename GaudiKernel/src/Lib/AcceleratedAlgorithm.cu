#include "Gaudi/CUDAAcceleratedAlgHelper.cuh"
#include <boost/fiber/cuda/waitfor.hpp>

#include <cstdio>
#include <cstring>
#include <tuple>

namespace Gaudi {
  namespace CUDA {
    const char* err_fmt( hipError_t err ) {
      if ( err == hipSuccess ) {
        return nullptr;
      } else {
        const char* errname = hipGetErrorName( err );
        const char* errstr  = hipGetErrorString( err );
        std::size_t msg_len = 28 + std::strlen( errname ) + std::strlen( errstr );
        char*       buf     = new char[msg_len];
        std::snprintf( buf, msg_len, "Encountered CUDA error %s: %s\n", errname, errstr );
        return buf;
      }
    }

    const char* create_stream( hipStream_t* stream_ptr ) { return err_fmt( hipStreamCreate( stream_ptr ) ); }

    const char* delete_stream( hipStream_t stream ) { return err_fmt( hipStreamDestroy( stream ) ); }

    const char* cuda_stream_await( hipStream_t cudaStream ) {
      auto        res = boost::fibers::cuda::waitfor_all( cudaStream );
      hipError_t err = std::get<1>( res );
      return err_fmt( err );
    }

  } // namespace CUDA
} // namespace Gaudi
