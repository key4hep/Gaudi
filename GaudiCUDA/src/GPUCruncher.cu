#include "hip/hip_runtime.h"
/***********************************************************************************\
* (c) Copyright 2023-2024 CERN for the benefit of the LHCb and ATLAS collaborations *
*                                                                                   *
* This software is distributed under the terms of the Apache version 2 licence,     *
* copied verbatim in the file "LICENSE".                                            *
*                                                                                   *
* In applying this licence, CERN does not waive the privileges and immunities       *
* granted to it by virtue of its status as an Intergovernmental Organization        *
* or submit itself to any jurisdiction.                                             *
\***********************************************************************************/

#include "GPUCruncher.h"

#include <Gaudi/CUDA/CUDAStream.h>

#include "CUDADeviceArray.h"

#include <cstdio>
#include <format>

#include <hipcub/hipcub.hpp>
#include <cub/util_device.cuh>
#define CUDA_CHECK( stmt )                                                                                              \
  if ( hipError_t cudaErr = hipGetLastError(); cudaErr != hipSuccess ) {                                             \
    const char* errname = hipGetErrorName( cudaErr );                                                                  \
    const char* errstr  = hipGetErrorString( cudaErr );                                                                \
    std::string errmsg  = std::format( "Encountered CUDA error BEFORE {} [{}]: {} on {}:{} (dev {})", errname,          \
                                       int( cudaErr ), errstr, __FILE__, __LINE__, hipcub::CurrentDevice() );              \
    error() << errmsg << endmsg;                                                                                        \
    return StatusCode::FAILURE;                                                                                         \
  }                                                                                                                     \
  if ( hipError_t cudaErr = ( stmt ); cudaErr != hipSuccess ) {                                                       \
    const char* errname = hipGetErrorName( cudaErr );                                                                  \
    const char* errstr  = hipGetErrorString( cudaErr );                                                                \
    std::string errmsg  = std::format( "Encountered CUDA error {} [{}]: {} on {}:{} (dev {})", errname, int( cudaErr ), \
                                       errstr, __FILE__, __LINE__, hipcub::CurrentDevice() );                              \
    error() << errmsg << endmsg;                                                                                        \
    return StatusCode::FAILURE;                                                                                         \
  }

__global__ void makeGrid( const double* input, double* workspace ) {
  double at                                        = ( threadIdx.x + 1 ) * input[blockIdx.x];
  workspace[blockIdx.x * blockDim.x + threadIdx.x] = at;
}

struct bounds_t {
  double lower = 0.;
  double upper = 1.;
};

StatusCode GPUCruncher::gpuExecute( const std::vector<double>& in, std::vector<double>& out ) const {
  CUDA_CHECK( hipSetDevice( 0 ) );
  // Create the stream
  Gaudi::CUDA::Stream stream( this );

  // Allocate device memory
  Gaudi::CUDA::DeviceArray<double> d_input( stream, in.size() );

  // Copy input
  d_input = in;

  // Run computation part 1 -- a math transform
  Gaudi::CUDA::DeviceArray<double> d_inter1( stream, 256 * in.size() ); // 256 doubles for each original double
  makeGrid<<<in.size(), 256, 0, stream>>>( &d_input, &d_inter1 );

  // Make histogram
  std::vector<int>                   hist( 10, 0 ); // histogram as ints
  Gaudi::CUDA::DeviceArray<int>      d_hist( stream, 10 );
  bounds_t                           bounds{};
  Gaudi::CUDA::DeviceArray<bounds_t> d_bounds( stream, 1 );

  std::size_t wkspace_size     = 0;
  std::size_t wkspace_size_tmp = 0;
  // determine workspace size
  CUDA_CHECK( hipcub::DeviceReduce::Min( nullptr, wkspace_size, &d_inter1, &d_bounds->lower, 256 * in.size(), stream ) );
  CUDA_CHECK(
      hipcub::DeviceReduce::Max( nullptr, wkspace_size_tmp, &d_inter1, &d_bounds->upper, 256 * in.size(), stream ) );
  if ( wkspace_size_tmp > wkspace_size ) wkspace_size = wkspace_size_tmp;
  wkspace_size_tmp = 0;
  CUDA_CHECK( hipcub::DeviceHistogram::HistogramEven( nullptr, wkspace_size_tmp, &d_inter1, &d_hist, hist.size() + 1,
                                                   bounds.lower, std::nextafter( bounds.upper, bounds.upper + 1 ),
                                                   256 * in.size(), stream ) );
  if ( wkspace_size_tmp > wkspace_size ) wkspace_size = wkspace_size_tmp;

  // Allocate workspace
  Gaudi::CUDA::DeviceArray<std::byte> d_wkspace( stream, wkspace_size );

  // Get bounds
  std::size_t temp = wkspace_size;
  CUDA_CHECK( hipcub::DeviceReduce::Min( &d_wkspace, temp, &d_inter1, &d_bounds->lower, 256 * in.size(), stream ) );
  temp = wkspace_size;
  CUDA_CHECK( hipcub::DeviceReduce::Max( &d_wkspace, temp, &d_inter1, &d_bounds->upper, 256 * in.size(), stream ) );
  // Copy bounds back to host. Implicit await while this happens
  d_bounds.toHost( bounds );

  // Make histogram
  temp = wkspace_size;
  for ( int i = 0; i < 10; ++i ) {
    CUDA_CHECK( hipcub::DeviceHistogram::HistogramEven( &d_wkspace, temp, &d_inter1, &d_hist, hist.size() + 1,
                                                     bounds.lower, std::nextafter( bounds.upper, bounds.upper + 1 ),
                                                     256 * in.size(), stream ) );
  }
  // Retrieve histogram. Implicit await while this happens.
  d_hist.toHost( hist );

  out = { bounds.lower, bounds.upper };
  out.reserve( hist.size() + 2 );
  for ( int x : hist ) { out.push_back( double( x ) ); }

  // Stream associated DeviceArrays get deallocated, then an implicit Stream await when the Stream is destroyed
  return StatusCode::SUCCESS;
}
