#include "hip/hip_runtime.h"
/***********************************************************************************\
* (c) Copyright 2023-2024 CERN for the benefit of the LHCb and ATLAS collaborations *
*                                                                                   *
* This software is distributed under the terms of the Apache version 2 licence,     *
* copied verbatim in the file "LICENSE".                                            *
*                                                                                   *
* In applying this licence, CERN does not waive the privileges and immunities       *
* granted to it by virtue of its status as an Intergovernmental Organization        *
* or submit itself to any jurisdiction.                                             *
\***********************************************************************************/

#include "GPUCruncher.h"

#include <cstdio>
#include <memory_resource>

#include <hipcub/hipcub.hpp>
__global__ void makeGrid( const double* input, double* workspace ) {
  double at                                        = threadIdx.x * input[blockIdx.x];
  workspace[blockIdx.x * blockDim.x + threadIdx.x] = at;
}

StatusCode GPUCruncher::gpuExecute( const std::pmr::vector<double>& in, std::vector<double>& out ) const {
  // Create the stream
  Gaudi::CUDA::CUDAStream stream( dynamic_cast<const Gaudi::AsynchronousAlgorithm*>( this ) );

  // Allocate device memory
  double* d_input = stream.malloc<double>( in.size() );

  // Copy input
  CUDA_CHECK( hipMemcpyAsync( d_input, in.data(), in.size() * sizeof( double ), hipMemcpyHostToDevice, stream ) );

  // Run computation part 1 -- a math transform
  double* d_inter1 = stream.malloc<double>( 256 * in.size() ); // 256 doubles for each original double
  makeGrid<<<in.size(), 256, 0, stream>>>( d_input, d_inter1 );

  // Make histogram
  std::pmr::vector<int> hist( in.size() - 2, 0, pinned ); // histogram as ints
  int*                  d_hist   = stream.malloc<int>( in.size() - 2 );
  double*               d_bounds = stream.malloc<double>( 2 );
  struct bounds_t {
    double lower = 0.;
    double upper = 1.;
  };
  bounds_t* bounds = static_cast<bounds_t*>( pinned->allocate( sizeof( bounds_t ) ) );

  void*       d_wkspace        = nullptr;
  std::size_t wkspace_size     = 0;
  std::size_t wkspace_size_tmp = 0;
  // determine workspace size
  CUDA_CHECK( hipcub::DeviceReduce::Min( d_wkspace, wkspace_size, d_inter1, d_bounds, 256 * in.size(), stream ) );
  CUDA_CHECK( hipcub::DeviceReduce::Max( d_wkspace, wkspace_size_tmp, d_inter1, d_bounds + 1, 256 * in.size(), stream ) );
  if ( wkspace_size_tmp > wkspace_size ) wkspace_size = wkspace_size_tmp;
  wkspace_size_tmp = 0;
  CUDA_CHECK( hipcub::DeviceHistogram::HistogramEven( d_wkspace, wkspace_size_tmp, d_inter1, d_hist, hist.size() + 1,
                                                   bounds->lower, bounds->upper, 256 * in.size(), stream ) );
  if ( wkspace_size_tmp > wkspace_size ) wkspace_size = wkspace_size_tmp;

  // Allocate workspace
  d_wkspace = stream.malloc<void>( wkspace_size );

  // Get bounds
  std::size_t temp = wkspace_size;
  CUDA_CHECK( hipcub::DeviceReduce::Min( d_wkspace, temp, d_inter1, d_bounds, 256 * in.size(), stream ) );
  temp = wkspace_size;
  CUDA_CHECK( hipcub::DeviceReduce::Max( d_wkspace, temp, d_inter1, d_bounds + 1, 256 * in.size(), stream ) );
  CUDA_CHECK( hipMemcpyAsync( bounds, d_bounds, 2 * sizeof( double ), hipMemcpyDeviceToHost, stream ) );
  // Sync so we have the bounds on the host
  StatusCode sc = cuda_stream_await( stream );
  if ( sc.isFailure() ) return sc;
  // Make histogram
  temp = wkspace_size;
  for ( int i = 0; i < 10; ++i ) {
    CUDA_CHECK( hipcub::DeviceHistogram::HistogramEven( d_wkspace, temp, d_inter1, d_hist, hist.size() + 1, bounds->lower,
                                                     bounds->upper, 256 * in.size(), stream ) );
  }
  // Retrieve histogram
  CUDA_CHECK( hipMemcpyAsync( hist.data(), d_hist, hist.size() * sizeof( int ), hipMemcpyDeviceToHost, stream ) );

  // Convert histogram to doubles
  out = { bounds->lower, bounds->upper };
  pinned->deallocate( bounds, sizeof( bounds_t ) );
  out.reserve( hist.size() + 2 );
  for ( int x : hist ) { out.push_back( double( x ) ); }
  return StatusCode::SUCCESS;
}
